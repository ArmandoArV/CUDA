#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void addArrays(int* a, int* b, int* c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;
    int* h_a, * h_b, * h_c;
    int* d_a, * d_b, * d_c;
    size_t size = n * sizeof(int);

    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    addArrays << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, n);
    hipEventRecord(stop);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time taken by the kernel: %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
